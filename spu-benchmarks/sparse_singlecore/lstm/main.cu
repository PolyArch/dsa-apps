#include <cstdio>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>

#include <math.h>

#define batch 1

#include <sys/time.h>

static __inline__ uint64_t rdtsc(void) {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (((uint64_t)tv.tv_sec) * 1000000 + ((uint64_t)tv.tv_usec));
}

static uint64_t ticks;

static void begin_roi() {

  ticks=rdtsc();

}


static void end_roi()   {

  ticks=(rdtsc()-ticks);
  printf("ticks: %lu\n", ticks);

}


__global__
void c_next(float *a, float *b, float *c, float *d, float *e) {
  int i = threadIdx.x;
  a[i] = 1.0 / (1.0 + exp(a[i]));
  c[i] = 1.0 / (1.0 + exp(c[i]));
  d[i] = 1.0 / (1.0 + exp(d[i]));
  e[i] = a[i] * b[i] + c[i] * d[i];
}

__global__
void h_next(float *a, float *b, float *c) {
  int i = blockIdx.x;
  c[i] = a[i] * tanh(b[i]);
}

__global__
void exp(float *a, float *sum) {
  int i = blockIdx.x;
  a[i] = exp(a[i]);
  sum[0] += a[i];
}

__global__
void div(float *a, float *sum) {
  int i = threadIdx.x;
  a[i] /= sum[0];
}

int main(int argc, char **argv) {
  int N;
  if (argc != 2) {
    N = 1024;
  } else {
    N = atoi(argv[1]);
  }

  hipblasHandle_t cu_handle;
  hipblasCreate(&cu_handle);

  float *x;
  hipMalloc(&x, N * sizeof(float));

  float *w_f, *b_f, *_f;
  hipMalloc(&w_f, N * N / 2 * sizeof(float));
  hipMalloc(&b_f, N / 2 * sizeof(float));
  hipMalloc(&_f, N / 2 * sizeof(float) * batch);

  float *w_i, *b_i, *_i;
  hipMalloc(&w_i, N * N / 2 * sizeof(float));
  hipMalloc(&b_i, N / 2 * sizeof(float));
  hipMalloc(&_i, N / 2 * sizeof(float) * batch);

  float *w_c, *b_c, *_c;
  hipMalloc(&w_c, N * N / 2 * sizeof(float));
  hipMalloc(&b_c, N / 2 * sizeof(float));
  hipMalloc(&_c, N / 2 * sizeof(float) * batch);

  float *w_o, *b_o, *_o;
  hipMalloc(&w_o, N * N / 2 * sizeof(float));
  hipMalloc(&b_o, N / 2 * sizeof(float));
  hipMalloc(&_o, N / 2 * sizeof(float) * batch);

  float *w_y, *b_y, *_y;
  hipMalloc(&w_y, N * N / 2 * sizeof(float));
  hipMalloc(&b_y, N / 2 * sizeof(float));
  hipMalloc(&_y, N / 2 * sizeof(float) * batch);

  float *c_nxt;
  hipMalloc(&c_nxt, N / 2 * sizeof(float) * batch);

  float *h_nxt;
  hipMalloc(&h_nxt, N / 2 * sizeof(float) * batch);

  float *sum;
  hipMalloc(&sum, sizeof(float));

  float alpha = 1.0;

  hipblasGemmEx(
    cu_handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_N,
    N / 2, N, 1,
    &alpha, w_f, HIP_R_32F, N,
    x, HIP_R_32F, 1,
    &alpha, b_f, HIP_R_32F, N,
    HIP_R_32F,
    HIPBLAS_GEMM_DEFAULT
  );

  hipMemcpy(_f, b_f, sizeof(float) * N / 2, hipMemcpyDefault);

  c_next<<<1, N / 2>>>(_f, x, _i, _c, c_nxt);
  exp<<<1, N / 2>>>(_y, sum);
  div<<<1, N / 2>>>(_y, sum);

  begin_roi();

  for (int i = 0; i < batch; ++i) {
    hipMemcpy(_f + i * N / 2, b_f, sizeof(float) * N / 2, hipMemcpyDefault);
    hipMemcpy(_i + i * N / 2, b_i, sizeof(float) * N / 2, hipMemcpyDefault);
    hipMemcpy(_c + i * N / 2, b_c, sizeof(float) * N / 2, hipMemcpyDefault);
    hipMemcpy(_o + i * N / 2, b_o, sizeof(float) * N / 2, hipMemcpyDefault);
    hipMemcpy(_y + i * N / 2, b_y, sizeof(float) * N / 2, hipMemcpyDefault);
  }

#define cublas_mv(m, a, b) \
  hipblasGemmEx( \
    cu_handle, \
    HIPBLAS_OP_N, \
    HIPBLAS_OP_N, \
    N / 2, N, batch, \
    &alpha, m, HIP_R_32F, N, \
    a, HIP_R_32F, batch, \
    &alpha, b, HIP_R_32F, N, \
    HIP_R_32F, \
    HIPBLAS_GEMM_DEFAULT \
  )

  cublas_mv(w_f, x, _f);
  cublas_mv(w_i, x, _i);
  cublas_mv(w_c, x, _c);
  cublas_mv(w_o, x, _o);

  c_next<<<N / 2 * batch, 1>>>(_f, x, _i, _c, c_nxt);
  h_next<<<N / 2 * batch, 1>>>(_o, c_nxt, h_nxt);

  hipblasGemmEx(
    cu_handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_N,
    N / 2, N / 2, batch,
    &alpha, w_y, HIP_R_32F, N / 2,
    h_nxt, HIP_R_32F, batch,
    &alpha, _y, HIP_R_32F, N / 2,
    HIP_R_32F,
    HIPBLAS_GEMM_DEFAULT
  );

  exp<<<N / 2 * batch, 1>>>(_y, sum);
  div<<<N / 2 * batch, 1>>>(_y, sum);

  end_roi();

  hipblasDestroy(cu_handle);

  return 0;
}
